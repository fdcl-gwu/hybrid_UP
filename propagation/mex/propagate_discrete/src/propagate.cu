#include "hip/hip_runtime.h"
#include "propagate.cuh"

#include <math.h>
#include <stdio.h>

void get_df_noise(myReal* df, const myReal* f, const myReal* lambda, const myReal* fcL, const int numR, const int* indR, const short* fcL_indx, const int nD, const Size_f* size_f)
{
    // set up fold
    myReal* f_dev;
    cudaErrorHandle(hipMalloc(&f_dev, size_f->nTot*sizeof(myReal)));
    cudaErrorHandle(hipMemcpy(f_dev, f, size_f->nTot*sizeof(myReal), hipMemcpyHostToDevice));

    int* indR_dev;
    cudaErrorHandle(hipMalloc(&indR_dev, numR*sizeof(int)));
    cudaErrorHandle(hipMemcpy(indR_dev, indR, numR*sizeof(int), hipMemcpyHostToDevice));

    short* fcL_indx_dev;
    cudaErrorHandle(hipMalloc(&fcL_indx_dev, nD*numR*size_f->nx*sizeof(short)));
    cudaErrorHandle(hipMemcpy(fcL_indx_dev, fcL_indx, nD*numR*size_f->nx*sizeof(short), hipMemcpyHostToDevice));

    Size_f* size_f_dev;
    cudaErrorHandle(hipMalloc(&size_f_dev, sizeof(Size_f)));
    cudaErrorHandle(hipMemcpy(size_f_dev, size_f, sizeof(Size_f), hipMemcpyHostToDevice));

    myReal* fold_dev;
    cudaErrorHandle(hipMalloc(&fold_dev, nD*numR*size_f->nx*sizeof(myReal)));

    dim3 blocksize_fcL(nD, size_f->const_2Bx, 1);
    dim3 gridsize_fcL(size_f->const_2Bx, numR, 1);

    get_fold_noise <<<gridsize_fcL, blocksize_fcL>>> (fold_dev, f_dev, indR_dev, fcL_indx_dev, size_f_dev);

    // calculate fin
    myReal* fcL_dev;
    cudaErrorHandle(hipMalloc(&fcL_dev, nD*numR*size_f->nx*sizeof(myReal)));
    cudaErrorHandle(hipMemcpy(fcL_dev, fcL, nD*numR*size_f->nx*sizeof(myReal), hipMemcpyHostToDevice));

    myReal* fin_dev;
    cudaErrorHandle(hipMalloc(&fin_dev, numR*size_f->nx*sizeof(myReal)));

    hiptensorHandle_t handle;
    cutensorInit(&handle);

    int32_t mode_fold[3] = {'x','R','y'};
    int32_t mode_fcL[3] = {'x','R','y'};
    int32_t mode_fin[2] = {'R','y'};

    int64_t extent_fold[3] = {nD, numR, size_f->nx};
    int64_t extent_fcL[3] = {nD, numR, size_f->nx};
    int64_t extent_fin[2] = {numR, size_f->nx};

    hiptensorTensorDescriptor_t desc_fold;
    hiptensorTensorDescriptor_t desc_fcL;
    hiptensorTensorDescriptor_t desc_fin;
    cutensorErrorHandle(hiptensorInitTensorDescriptor(&handle, &desc_fold, 3, extent_fold, NULL, mycudaRealType, HIPTENSOR_OP_IDENTITY));
    cutensorErrorHandle(hiptensorInitTensorDescriptor(&handle, &desc_fcL, 3, extent_fcL, NULL, mycudaRealType, HIPTENSOR_OP_IDENTITY));
    cutensorErrorHandle(hiptensorInitTensorDescriptor(&handle, &desc_fin, 2, extent_fin, NULL, mycudaRealType, HIPTENSOR_OP_IDENTITY));

    uint32_t alignment_fold;
    uint32_t alignment_fcL;
    uint32_t alignment_fin;
    cutensorErrorHandle(cutensorGetAlignmentRequirement(&handle, fold_dev, &desc_fold, &alignment_fold));
    cutensorErrorHandle(cutensorGetAlignmentRequirement(&handle, fcL_dev, &desc_fcL, &alignment_fcL));
    cutensorErrorHandle(cutensorGetAlignmentRequirement(&handle, fin_dev, &desc_fin, &alignment_fin));

    cutensorContractionDescriptor_t desc;
    cutensorErrorHandle(cutensorInitContractionDescriptor(&handle, &desc, &desc_fold, mode_fold, alignment_fold,
        &desc_fcL, mode_fcL, alignment_fcL,
        &desc_fin, mode_fin, alignment_fin,
        &desc_fin, mode_fin, alignment_fin, mycutensor_computetype));

    cutensorContractionFind_t find;
    cutensorErrorHandle(cutensorInitContractionFind(&handle, &find, HIPTENSOR_ALGO_DEFAULT));

    size_t worksize;
    cutensorErrorHandle(cutensorContractionGetWorkspace(&handle, &desc, &find, CUTENSOR_WORKSPACE_RECOMMENDED, &worksize));
    void* workspace = nullptr;
    if (worksize > 0) {
        cudaErrorHandle(hipMalloc(&workspace, worksize));
    }

    hiptensorContractionPlan_t plan;
    cutensorErrorHandle(cutensorInitContractionPlan(&handle, &plan, &desc, &find, worksize));

    myReal alpha = 1.0;
    myReal beta = 0.0;

    cutensorErrorHandle(hiptensorContraction(&handle, &plan, &alpha, fold_dev, fcL_dev, &beta, fin_dev, fin_dev, workspace, worksize, 0));

    // free memory
    cudaErrorHandle(hipFree(fcL_indx_dev));
    cudaErrorHandle(hipFree(fold_dev));
    cudaErrorHandle(hipFree(fcL_dev));

    if (worksize > 0) {
        cudaErrorHandle(hipFree(workspace));
    }

    // compute fout
    myReal* df_dev;
    cudaErrorHandle(hipMalloc(&df_dev, size_f->nTot*sizeof(myReal)));
    cudaErrorHandle(hipMemset(df_dev, 0, size_f->nTot*sizeof(myReal)));

    myReal* lambda_dev;
    cudaErrorHandle(hipMalloc(&lambda_dev, numR*size_f->nx*sizeof(myReal)));
    cudaErrorHandle(hipMemcpy(lambda_dev, lambda, numR*size_f->nx*sizeof(myReal), hipMemcpyHostToDevice));

    dim3 blocksize_n0Rx(size_f->const_2Bx, 1, 1);
    dim3 gridsize_n0Rx(size_f->const_2Bx, numR, 1);
    
    get_fout_noise <<<gridsize_n0Rx, blocksize_n0Rx>>> (df_dev, fin_dev, f_dev, lambda_dev, indR_dev, size_f_dev);
    cudaErrorHandle(hipMemcpy(df, df_dev, size_f->nTot*sizeof(myReal), hipMemcpyDeviceToHost));

    // free memory
    cudaErrorHandle(hipFree(f_dev));
    cudaErrorHandle(hipFree(indR_dev));
    cudaErrorHandle(hipFree(size_f_dev));
    cudaErrorHandle(hipFree(fin_dev));
    cudaErrorHandle(hipFree(df_dev));
    cudaErrorHandle(hipFree(lambda_dev));
}

void get_df_nonoise(myReal* df, const myReal* f, const myReal* lambda, const int numR, const int* indR, int* const* lambda_indx, const int* lambda_numx, const int* ind_interp, const myReal* coeff_interp, const Size_f* size_f)
{
    // compute fin
    dim3 blocksize_n0Rx(size_f->const_2Bx, 1, 1);
    dim3 gridsize_n0Rx(size_f->const_2Bx, numR, 1);

    myReal* f_dev;
    cudaErrorHandle(hipMalloc(&f_dev, size_f->nTot*sizeof(myReal)));
    cudaErrorHandle(hipMemcpy(f_dev, f, size_f->nTot*sizeof(myReal), hipMemcpyHostToDevice));

    myReal* lambda_dev;
    cudaErrorHandle(hipMalloc(&lambda_dev, numR*sizeof(myReal)));
    cudaErrorHandle(hipMemcpy(lambda_dev, lambda, numR*sizeof(myReal), hipMemcpyHostToDevice));

    int* indR_dev;
    cudaErrorHandle(hipMalloc(&indR_dev, numR*sizeof(int)));
    cudaErrorHandle(hipMemcpy(indR_dev, indR, numR*sizeof(int), hipMemcpyHostToDevice));

    int* ind_interp_dev;
    cudaErrorHandle(hipMalloc(&ind_interp_dev, 4*numR*size_f->nx*sizeof(int)));
    cudaErrorHandle(hipMemcpy(ind_interp_dev, ind_interp, 4*numR*size_f->nx*sizeof(int), hipMemcpyHostToDevice));
    
    myReal* coeff_interp_dev;
    cudaErrorHandle(hipMalloc(&coeff_interp_dev, 4*numR*size_f->nx*sizeof(myReal)));
    cudaErrorHandle(hipMemcpy(coeff_interp_dev, coeff_interp, 4*numR*size_f->nx*sizeof(myReal), hipMemcpyHostToDevice));
    
    Size_f* size_f_dev;
    cudaErrorHandle(hipMalloc(&size_f_dev, sizeof(Size_f)));
    cudaErrorHandle(hipMemcpy(size_f_dev, size_f, sizeof(Size_f), hipMemcpyHostToDevice));

    myReal* df_dev;
    cudaErrorHandle(hipMalloc(&df_dev, size_f->nTot*sizeof(myReal)));
    cudaErrorHandle(hipMemset(df_dev, 0, size_f->nTot*sizeof(myReal)));

    get_fin_nonoise <<<gridsize_n0Rx, blocksize_n0Rx>>> (df_dev, f_dev, lambda_dev, indR_dev, ind_interp_dev, coeff_interp_dev, size_f_dev);    

    // compute fout
    int* lambda_indx_dev;
    cudaErrorHandle(hipMalloc(&lambda_indx_dev, size_f->nx*sizeof(int)));

    for (int iR = 0; iR < numR; iR++) {
        cudaErrorHandle(hipMemcpy(lambda_indx_dev, lambda_indx[iR], lambda_numx[iR]*sizeof(int), hipMemcpyHostToDevice));
        get_fout_nonoise <<<(int)lambda_numx[iR]/128+1, 128>>> (df_dev+indR[iR], f_dev+indR[iR], lambda[iR], lambda_indx_dev, lambda_numx[iR], size_f_dev);
    }

    cudaErrorHandle(hipMemcpy(df, df_dev, size_f->nTot*sizeof(myReal), hipMemcpyDeviceToHost));

    // free memory
    cudaErrorHandle(hipFree(f_dev));
    cudaErrorHandle(hipFree(lambda_dev));
    cudaErrorHandle(hipFree(indR_dev));
    cudaErrorHandle(hipFree(ind_interp_dev));
    cudaErrorHandle(hipFree(coeff_interp_dev));
    cudaErrorHandle(hipFree(size_f_dev));
    cudaErrorHandle(hipFree(df_dev));
    cudaErrorHandle(hipFree(lambda_indx_dev));
}

__global__ void get_fold_noise(myReal* f_old, const myReal* f, const int* indR, const short* fcL_indx, const Size_f* size_f)
{
    int ind_fcL = threadIdx.x + blockIdx.y*blockDim.x + (threadIdx.y+blockIdx.x*blockDim.y)*blockDim.x*gridDim.y;
    int ind_f = indR[blockIdx.y] + fcL_indx[ind_fcL]*size_f->nR;

    f_old[ind_fcL] = f[ind_f];
}

__global__ void get_fin_nonoise(myReal* df, const myReal* f, const myReal* lambda, const int* indR, const int* ind_interp, const myReal* coeff_interp, const Size_f* size_f)
{
    int indx = threadIdx.x + blockIdx.x*blockDim.x;
    int indfR = indR[blockIdx.y];
    int indf = indfR + indx*size_f->nR;
    int indInterp = 4*(blockIdx.y + indx*gridDim.y);

    if (isnan(coeff_interp[indInterp])) {
        df[indf] = 0;
    } else {
        int indf_interp[4];
        for (int i = 0; i < 4; i++) {
            indf_interp[i] = indfR + ind_interp[indInterp+i]*size_f->nR;
        }

        myReal f_interp = 0.0;
        for (int i = 0; i < 4; i++) {
            f_interp += f[indf_interp[i]]*coeff_interp[indInterp+i];
        }

        df[indf] = f_interp*lambda[blockIdx.y];
    }
}

__global__ void get_fout_noise(myReal* df, const myReal* fin, const myReal* f, const myReal* lambda, const int* indR, const Size_f* size_f)
{
    int indx = threadIdx.x + blockIdx.x*blockDim.x;
    int indf = indR[blockIdx.y] + indx*size_f->nR;
    int indfin = blockIdx.y + indx*gridDim.y;

    df[indf] = fin[indfin] - f[indf]*lambda[indfin];
}

__global__ void get_fout_nonoise(myReal* df, const myReal* f, const myReal lambda, const int* lambda_indx, const int lambda_numx, const Size_f* size_f)
{
    int indx = threadIdx.x + blockIdx.x*blockDim.x;
    if (indx < lambda_numx) {
        int indf = lambda_indx[indx]*size_f->nR;
        df[indf] = df[indf] - f[indf]*lambda;
    }
}


