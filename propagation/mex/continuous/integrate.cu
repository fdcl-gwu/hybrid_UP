#include "hip/hip_runtime.h"
#include "integrate.cuh"

#include <stdio.h>
#include <iostream>

#undef printf

__global__ void flip_shift(const myComplex* X, myComplex* X_ij, const int is, const int js, const Size_F* size_F)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int j = blockIdx.y*blockDim.y + threadIdx.y;

	if (i < size_F[0].const_2Bx && j < size_F[0].const_2Bx) {
		int iout = is-i;
		if (iout < 0)
			iout += size_F[0].const_2Bx;
		else if (iout >= size_F[0].const_2Bx)
			iout -= size_F[0].const_2Bx;

		int jout = js-j;
		if (jout < 0)
			jout += size_F[0].const_2Bx;
		else if (jout >= size_F[0].const_2Bx)
			jout -= size_F[0].const_2Bx;

		int X_ind = i + j*size_F[0].const_2Bx;
		int X_ij_ind = iout + jout*size_F[0].const_2Bx;

		for (int ip = 0; ip < 2; ip++)
			X_ij[X_ij_ind + ip*size_F[0].nx] = X[X_ind + ip*size_F[0].nx];
	}
}

__global__ void addup_F(myComplex* dF, const int nTot)
{
	int ind1 = threadIdx.x + blockIdx.x*blockDim.x;
	if (ind1 < nTot) {
		int ind2 = ind1 + nTot;
		dF[ind1] = mycuCadd(dF[ind1], dF[ind2]);
	}
}

__global__ void add_F(myComplex* dF, const myComplex* dF_temp, const int nTot)
{
	int ind = threadIdx.x + blockIdx.x*blockDim.x;
	if (ind < nTot)
		dF[ind] = mycuCadd(dF[ind], dF_temp[ind]);
}

__global__ void mulImg_FR(myComplex* dF, const myReal c, const int nR)
{
	int ind = threadIdx.x + blockIdx.x*blockDim.x;
	if (ind < nR) {
		myReal y = dF[ind].y;
		dF[ind].y = dF[ind].x * c;
		dF[ind].x = -y * c;
	}
}

__global__ void add_FMR(myComplex* dF, const myComplex* FMR, const int ind_cumR, const Size_F* size_F)
{
	int ind_dF = ind_cumR + (threadIdx.x + threadIdx.y*size_F->const_2Bx)*size_F->nR_compact + blockIdx.x*size_F->nTot_compact;
	int ind_FMR = threadIdx.x + threadIdx.y*size_F->const_2Bx + blockIdx.x*size_F->nx;

	dF[ind_dF] = mycuCadd(dF[ind_dF], FMR[ind_FMR]);
}

__global__ void mul_fmR(myReal* f, const myReal* mR, const int dim, const Size_f* size_f)
{
	int ind_F = threadIdx.x + blockIdx.x*blockDim.x;
	if (ind_F < size_f->nR) {
		int ind_mR = dim + ind_F*2;
		ind_F += blockIdx.y*size_f->nR;
		
		f[ind_F] = f[ind_F] * mR[ind_mR];
	}
}

__global__ void mulImg_FTot(myComplex* dF, const myReal* c, const int dim, const Size_F* size_F)
{
	int ind_R = threadIdx.x + blockIdx.x*blockDim.x;
	if (ind_R < size_F->nR_compact) {
		unsigned int ij[2] = {blockIdx.y, blockIdx.z};
		int ind_dF = ind_R + (ij[0] + ij[1]*size_F->const_2Bx)*size_F->nR_compact;

		myReal y = dF[ind_dF].y;
		dF[ind_dF].y = dF[ind_dF].x * c[ij[dim]];
		dF[ind_dF].x = -y * c[ij[dim]];
	}
}

__global__ void get_c(myReal* c, const int i, const int j, const myReal* L, const myReal* G, const Size_F* size_F)
{
	if (i == j) {
		int ix = threadIdx.x;
		if (ix < size_F[0].Bx)
			c[ix] = -4*PI*PI * ix*ix * G[i+2*j] / (L[0]*L[0]);
		else
			c[ix] = -4*PI*PI * (ix-size_F[0].const_2Bx)*(ix-size_F[0].const_2Bx) * G[i+2*j] / (L[0]*L[0]);
	} else {
		int ix = threadIdx.x;
		int jx = blockIdx.x;

		myReal c1;
		if (ix < size_F[0].Bx)
			c1 = 2*PI * ix / L[0];
		else if (ix == size_F[0].Bx)
			c1 = 0;
		else
			c1 = 2*PI * (ix-size_F[0].const_2Bx) / L[0];

		myReal c2;
		if (jx < size_F[0].Bx)
			c2 = 2*PI * jx / L[0];
		else if (jx == size_F[0].Bx)
			c2 = 0;
		else
			c2 = 2*PI * (jx-size_F[0].const_2Bx) / L[0];

		int indc = ix + jx*size_F[0].const_2Bx;
		c[indc] = -c1*c2 * G[i+2*j];
	}
}

__global__ void get_biasRW(myComplex* dF_temp, const myComplex* Fold, const myReal* c, const int i, const int j, const Size_F* size_F)
{
	int indR = threadIdx.x + blockIdx.x*blockDim.x;
	if (indR < size_F[0].nR_compact) {
		unsigned int ij[2] = {blockIdx.y, blockIdx.z};

		int ind = indR + (ij[0] + ij[1]*size_F->const_2Bx)*size_F[0].nR_compact;

		if (i==j) {
			dF_temp[ind].x = Fold[ind].x * c[ij[i]];
			dF_temp[ind].y = Fold[ind].y * c[ij[i]];
		} else {
			int indc = ij[i] + ij[j]*size_F[0].const_2Bx;
			dF_temp[ind].x = Fold[ind].x * c[indc];
			dF_temp[ind].y = Fold[ind].y * c[indc];
		}
	}
}

__global__ void integrate_Fnew(myComplex* Fnew, const myComplex* Fold, const myComplex* dF, const myReal dt, const int nTot)
{
	int ind = threadIdx.x + blockIdx.x*blockDim.x;
	if (ind < nTot)
	{
		Fnew[ind].x = Fold[ind].x + dt*dF[ind].x;
		Fnew[ind].y = Fold[ind].y + dt*dF[ind].y;
	}
}

__host__ void modify_F(const myComplex* F, myComplex* F_modify, bool reduce, Size_F* size_F)
{
	if (reduce) {
		int ind_F_reduced = 0;
		for (int j = 0; j < size_F[0].const_2Bx; j++) {
			for (int i = 0; i < size_F[0].const_2Bx; i++) {
				for (int l = 0; l <= size_F[0].lmax; l++) {
					for (int m = -l; m <= l; m++) {
						for (int n = -l; n <= l; n++) {
							int ind_F = n+size_F[0].lmax + (m+size_F[0].lmax)*size_F[0].l_cum0 + 
								l*size_F[0].l_cum1 + i*size_F[0].l_cum2 + j*size_F[0].l_cum3;
							F_modify[ind_F_reduced] = F[ind_F];

							ind_F_reduced++;
						}
					}
				}
			}
		}
	} else {
		int ind_F_reduced = 0;
		for (int j = 0; j < size_F[0].const_2Bx; j++) {
			for (int i = 0; i < size_F[0].const_2Bx; i++) {
				for (int l = 0; l <= size_F[0].lmax; l++) {
					for (int m = -l; m <= l; m++) {
						for (int n = -l; n <= l; n++) {
							int ind_F = n+size_F[0].lmax + (m+size_F[0].lmax)*size_F[0].l_cum0 + 
								l*size_F[0].l_cum1 + i*size_F[0].l_cum2 + j*size_F[0].l_cum3;
							F_modify[ind_F] = F[ind_F_reduced];

							ind_F_reduced++;
						}
					}
				}
			}
		}
	}
}

__host__ void modify_u(const myComplex* u, myComplex* u_modify, Size_F* size_F)
{
	int ind_u_reduced = 0;
	for (int ip = 0; ip < 2; ip++) {
		for (int l = 0; l <= size_F[0].lmax; l++) {
			for (int m = -l; m <= l; m++) {
				for (int n = -l; n <= l; n++) {
					int ind_u = n+size_F[0].lmax + (m+size_F[0].lmax)*size_F[0].l_cum0 + l*size_F[0].l_cum1 + ip*size_F[0].l_cum2;
					u_modify[ind_u_reduced] = u[ind_u];

					ind_u_reduced++;
				}
			}
		}
	}
}

__host__ void deriv_x(myReal* c, const int n, const int B, const myReal L)
{
	if (n < B)
		*c = 2*PI*n/L;
	else if (n == B)
		*c = 0;
	else
		*c = 2*PI*(n-2*B)/L;
}

__host__ void get_dF(myComplex* dF, const myComplex* F, const myReal* f, const myComplex* X, const myReal* mR, const myReal* b, const myReal* G,
	const myReal* L, const myComplex* u, const myReal* dw_dev, const Size_F* size_F, const Size_F* size_F_dev, const Size_f* size_f, const Size_f* size_f_dev)
{
	////////////////////////////
	// circular_convolution X //
	////////////////////////////

	// X_ijk = flip(flip(flip(X,1),2),3)
	// X_ijk = circshift(X_ijk,1,i)
	// X_ijk = circshift(X_ijk,2,j)
	// X_ijk = circshift(X_ijk,3,k)
	// dF{r,i,j,k,p} = F{r,m,n,l}.*X_ijk{m,n,l,p}
	// dF(indmn,indmn,l,i,j,k,p) = -dF(indmn,indmn,l,i,j,k,p)*u(indmn,indmn,l,p)'
	// dF = sum(dF,'p')

	// set up arrays
	myComplex* F_dev;
	cudaErrorHandle(hipMalloc(&F_dev, size_F->nTot_compact*sizeof(myComplex)));
	cudaErrorHandle(hipMemcpy(F_dev, F, size_F->nTot_compact*sizeof(myComplex), hipMemcpyHostToDevice));

	myComplex* X_dev;
	cudaErrorHandle(hipMalloc(&X_dev, 2*size_F->nx*sizeof(myComplex)));
	cudaErrorHandle(hipMemcpy(X_dev, X, 2*size_F->nx*sizeof(myComplex), hipMemcpyHostToDevice));

	myComplex* X_ij_dev;
	cudaErrorHandle(hipMalloc(&X_ij_dev, 2*size_F->nx*sizeof(myComplex)));

	myComplex* dF2_dev;
	cudaErrorHandle(hipMalloc(&dF2_dev, 2*size_F->nTot_compact*sizeof(myComplex)));

	myComplex* dF2_temp_dev;
	cudaErrorHandle(hipMalloc(&dF2_temp_dev, 2*size_F->nTot_compact*sizeof(myComplex)));

	myComplex* dF_temp_dev;
	cudaErrorHandle(hipMalloc(&dF_temp_dev, 2*size_F->nR_compact*sizeof(myComplex)));

	myComplex* u_dev;
	cudaErrorHandle(hipMalloc(&u_dev, 2*size_F->nR_compact*sizeof(myComplex)));
	cudaErrorHandle(hipMemcpy(u_dev, u, 2*size_F->nR_compact*sizeof(myComplex), hipMemcpyHostToDevice));

	myComplex* dF_dev;
	cudaErrorHandle(hipMalloc(&dF_dev, size_F->nTot_compact*sizeof(myComplex)));

	// set up cublas
	hipblasHandle_t handle_cublas;
	hipblasCreate(&handle_cublas);

	myComplex alpha_cublas = make_myComplex(1,0);
	myComplex beta_cublas = make_myComplex(0,0);

	// set up cutensor
	hiptensorHandle_t handle_cutensor;
	cutensorInit(&handle_cutensor);

	hiptensorContractionPlan_t plan_conv;
	size_t worksize_conv;

	cutensor_initConv(&handle_cutensor, &plan_conv, &worksize_conv, F_dev, X_ij_dev, dF_temp_dev, size_F);

	void* work = nullptr;
	if (worksize_conv > 0)
		cudaErrorHandle(hipMalloc(&work, worksize_conv));

	myComplex alpha_cutensor = make_myComplex(0-(myReal)1/size_F->nx,0);
	myComplex beta_cutensor = make_myComplex(0,0);

	// set up blocksize and gridsize
	dim3 blocksize_16(16, 16, 1);
	int gridnum_16 = (int) size_F->const_2Bx/16 + 1;
	dim3 gridsize_16(gridnum_16, gridnum_16, 1);

	dim3 blocksize_512_nTot(512, 1, 1);
	dim3 gridsize_512_nTot((int)size_F->nTot_compact/512+1, 1, 1);

	// calculate
	for (int i = 0; i < size_F->const_2Bx; i++) {
		for (int j = 0; j < size_F->const_2Bx; j++) {
			flip_shift <<<gridsize_16, blocksize_16>>> (X_dev, X_ij_dev, i, j, size_F_dev);
			cudaErrorHandle(hipGetLastError());

			cutensorErrorHandle(hiptensorContraction(&handle_cutensor, &plan_conv, (void*)&alpha_cutensor, F_dev, X_ij_dev,
				(void*)&beta_cutensor, dF_temp_dev, dF_temp_dev, work, worksize_conv, 0));

			for (int ip = 0; ip < 2; ip++) {
				myComplex* dF2_dev_ijn = dF2_dev + i*size_F->nR_compact + 
					j*(size_F->nR_compact*size_F->const_2Bx) + ip*size_F->nTot_compact;
				myComplex* dF_temp_dev_n = dF_temp_dev + ip*size_F->nR_compact;

				cudaErrorHandle(hipMemcpy(dF2_dev_ijn, dF_temp_dev_n, size_F->nR_compact*sizeof(myComplex), hipMemcpyDeviceToDevice));
			}
		}
	}

	for (int ip = 0; ip < 2; ip++) {
		for (int l = 0; l <= size_F->lmax; l++)
		{
			int ind_dF = l*(2*l-1)*(2*l+1)/3 + ip*size_F->nTot_compact;
			long long int stride_Fnew = size_F->nR_compact;

			int ind_u = l*(2*l-1)*(2*l+1)/3 + ip*size_F->nR_compact;
			long long int stride_u = 0;

			cublasErrorHandle(mycublasgemmStridedBatched(handle_cublas, HIPBLAS_OP_N, HIPBLAS_OP_T, 2*l+1, 2*l+1, 2*l+1,
				&alpha_cublas, dF2_dev+ind_dF, 2*l+1, stride_Fnew,
				u_dev+ind_u, 2*l+1, stride_u,
				&beta_cublas, dF2_temp_dev+ind_dF, 2*l+1, stride_Fnew, size_F->nx));
		}
	}

	addup_F <<<gridsize_512_nTot, blocksize_512_nTot>>> (dF2_temp_dev, size_F->nTot_compact);
	cudaErrorHandle(hipGetLastError());

	cudaErrorHandle(hipMemcpy(dF_dev, dF2_temp_dev, size_F->nTot_compact*sizeof(myComplex), hipMemcpyDeviceToDevice));

	// free memory
	cudaErrorHandle(hipFree(u_dev));
	cudaErrorHandle(hipFree(dF2_temp_dev));

	cublasErrorHandle(hipblasDestroy(handle_cublas));

	//////////////////////////////
	// circular convolutions bX //
	//////////////////////////////

	// bX_ijk = flip(flip(flip(-b*X,1),2),3)
	// bX_ijk = circshift(bX_ijk,1,i)
	// bX_ijk = circshift(bX_ijk,2,j)
	// bX_ijk = circshift(bX_ijk,3,k)
	// dF{r,i,j,k,p} = Fold{r,m,n,l}.*bX_ijk{m,n,l,p}
	// dF{r,i,j,k,p} = dF{r,i,j,k,p}*c(p)
	// dF = sum(dF,'p')

	// set up blocksize and gridsize
	dim3 blocksize_512_nR(512, 1, 1);
	dim3 gridsize_512_nR((int)size_F->nR_compact/512+1, 1, 1);

	// calculate
	for (int i = 0; i < size_F->const_2Bx; i++) {
		for (int j = 0; j < size_F->const_2Bx; j++) {
			flip_shift <<<gridsize_16, blocksize_16>>> (X_dev, X_ij_dev, i, j, size_F_dev);
			cudaErrorHandle(hipGetLastError());

			cutensorErrorHandle(hiptensorContraction(&handle_cutensor, &plan_conv, (void*)&alpha_cutensor, F_dev, X_ij_dev,
				(void*)&beta_cutensor, dF_temp_dev, dF_temp_dev, work, worksize_conv, 0));

			myReal c[2];
			deriv_x(c, i, size_F->Bx, *L);
			deriv_x(c+1, j, size_F->Bx, *L);

			mulImg_FR <<<gridsize_512_nR, blocksize_512_nR>>> (dF_temp_dev, -c[0]*b[0], size_F->nR_compact);
			cudaErrorHandle(hipGetLastError());
			mulImg_FR <<<gridsize_512_nR, blocksize_512_nR>>> (dF_temp_dev+size_F->nR_compact, -c[1]*b[1], size_F->nR_compact);
			cudaErrorHandle(hipGetLastError());

			for (int ip = 0; ip < 2; ip++) {
				myComplex* dF2_dev_ijp = dF2_dev + i*size_F->nR_compact + 
					j*(size_F->nR_compact*size_F->const_2Bx) + ip*size_F->nTot_compact;
				myComplex* dF_temp_dev_p = dF_temp_dev + ip*size_F->nR_compact;

				cudaErrorHandle(hipMemcpy(dF2_dev_ijp, dF_temp_dev_p, size_F->nR_compact*sizeof(myComplex), hipMemcpyDeviceToDevice));
			}
		}
	}

	addup_F <<<gridsize_512_nTot, blocksize_512_nTot>>> (dF2_dev, size_F->nTot_compact);
	cudaErrorHandle(hipGetLastError());

	add_F <<<gridsize_512_nTot, blocksize_512_nTot>>> (dF_dev, dF2_dev, size_F->nTot_compact);
	cudaErrorHandle(hipGetLastError());

	// free memory
	cudaErrorHandle(hipFree(X_dev));
	cudaErrorHandle(hipFree(X_ij_dev));
	cudaErrorHandle(hipFree(dF_temp_dev));
	if (worksize_conv > 0)
		cudaErrorHandle(hipFree(work));

	/////////////////
	// multiply mR //
	/////////////////

	// set up arrays
	myReal* fmR_dev;
	cudaErrorHandle(hipMalloc(&fmR_dev, size_f->nTot*sizeof(myReal)));
	cudaErrorHandle(hipMemcpy(fmR_dev, f, size_f->nTot*sizeof(myReal), hipMemcpyHostToDevice));

	myReal* mR_dev;
	cudaErrorHandle(hipMalloc(&mR_dev, 2*size_f->nR*sizeof(myReal)));
	cudaErrorHandle(hipMemcpy(mR_dev, mR, 2*size_f->nR*sizeof(myReal), hipMemcpyHostToDevice));

	// get c
	myReal* c = new myReal[size_F->const_2Bx];
	for (int i = 0; i < size_F->const_2Bx; i++) {
		deriv_x(&c[i], i, size_F->Bx, *L);
		c[i] = -c[i];
	}

	myReal* c_dev;
	cudaErrorHandle(hipMalloc(&c_dev, size_F->const_2Bx*sizeof(myReal)));
	cudaErrorHandle(hipMemcpy(c_dev, c, size_F->const_2Bx*sizeof(myReal), hipMemcpyHostToDevice));

	// set up blocksize and gridsize
	dim3 blocksize_512_nRf_nx(512, 1, 1);
	dim3 gridsize_512_nRf_nx((int)size_f->nR/512+1, size_f->nx, 1);

	dim3 blocksize_deriv(512,1,1);
	dim3 gridsize_deriv((int)size_F->nR_compact/512+1, size_F->const_2Bx, size_F->const_2Bx);

	// calculate
	mul_fmR <<<gridsize_512_nRf_nx, blocksize_512_nRf_nx>>> (fmR_dev, mR_dev, 0, size_f_dev);
	fftSO3R_forward(dF2_dev, fmR_dev, dw_dev, size_F, size_F_dev, size_f, size_f_dev);

	cudaErrorHandle(hipMemcpy(fmR_dev, f, size_f->nTot*sizeof(myReal), hipMemcpyHostToDevice));
	mul_fmR <<<gridsize_512_nRf_nx, blocksize_512_nRf_nx>>> (fmR_dev, mR_dev, 1, size_f_dev);
	fftSO3R_forward(dF2_dev+size_F->nTot_compact, fmR_dev, dw_dev, size_F, size_F_dev, size_f, size_f_dev);

	for (int ip = 0; ip < 2; ip++) {
		mulImg_FTot <<<gridsize_deriv, blocksize_deriv>>> (dF2_dev+ip*size_F->nTot_compact, c_dev, ip, size_F_dev);
		cudaErrorHandle(hipGetLastError());
	}

	addup_F <<<gridsize_512_nTot, blocksize_512_nTot>>> (dF2_dev, size_F->nTot_compact);
	cudaErrorHandle(hipGetLastError());

	add_F <<<gridsize_512_nTot, blocksize_512_nTot>>> (dF_dev, dF2_dev, size_F->nTot_compact);
	cudaErrorHandle(hipGetLastError());

	// free memory
	cudaErrorHandle(hipFree(fmR_dev));
	cudaErrorHandle(hipFree(mR_dev));
	cudaErrorHandle(hipFree(c_dev));

	delete[] c;

	///////////////////////
	// random walk noise //
	///////////////////////

	// set up arrays
	cudaErrorHandle(hipMalloc(&c_dev, size_F->nx*sizeof(myReal)));

	myReal* G_dev;
	cudaErrorHandle(hipMalloc(&G_dev, 9*sizeof(myReal)));
	cudaErrorHandle(hipMemcpy(G_dev, G, 9*sizeof(myReal), hipMemcpyHostToDevice));

	myReal* L_dev;
	cudaErrorHandle(hipMalloc(&L_dev, sizeof(myReal)));
	cudaErrorHandle(hipMemcpy(L_dev, L, sizeof(myReal), hipMemcpyHostToDevice));

	// calculate
	for (int i = 0; i < 2; i++) {
		for (int j = 0; j < 2; j++) {
			if (i == j) {
				get_c <<<1, size_F->const_2Bx>>> (c_dev, i, j, L_dev, G_dev, size_F_dev);
				cudaErrorHandle(hipGetLastError());
			}
			else {
				get_c <<<size_F->const_2Bx, size_F->const_2Bx>>> (c_dev, i, j, L_dev, G_dev, size_F_dev);
				cudaErrorHandle(hipGetLastError());
			}

			get_biasRW <<<gridsize_deriv, blocksize_deriv>>> (dF2_dev, F_dev, c_dev, i, j, size_F_dev);
			cudaErrorHandle(hipGetLastError());

			add_F <<<gridsize_512_nTot, blocksize_512_nTot>>> (dF_dev, dF2_dev, size_F->nTot_compact);
			cudaErrorHandle(hipGetLastError());
		}
	}

	// free memory
	cudaErrorHandle(hipFree(c_dev));
	cudaErrorHandle(hipFree(G_dev));
	cudaErrorHandle(hipFree(L_dev));
	cudaErrorHandle(hipFree(F_dev));
	cudaErrorHandle(hipFree(dF2_dev));

	// return
	cudaErrorHandle(hipMemcpy(dF, dF_dev, size_F->nTot_compact*sizeof(myComplex), hipMemcpyDeviceToHost));

	cudaErrorHandle(hipFree(dF_dev));
}

