#include "hip/hip_runtime.h"
#include "getFcL.cuh"

#include <stdio.h>
#include <math.h>

#include <thrust/sort.h>
#include <thrust/device_ptr.h>
#include <thrust/functional.h>
#include <thrust/reduce.h>

void getFcL(myReal* fcL, short* fcL_indx, const myReal* x, const myReal* Omega, const myReal* lambda, const int numR, const myReal* Gd, const int nD, const Size_f* size_f)
{
    // pre-calculations
    myReal detGd = Gd[0]*Gd[3] - Gd[2]*Gd[1];
    myReal c_normal = 1/(2*PI*mysqrt(detGd));

    myReal invGd[4];
    invGd[0] = Gd[3]/detGd;
    invGd[1] = -Gd[2]/detGd;
    invGd[2] = -Gd[1]/detGd;
    invGd[3] = Gd[0]/detGd;

    myReal dx2 = (x[2]-x[0]) * (x[2]-x[0]);

    // calculate fc
    myReal* x_dev;
    cudaErrorHandle(hipMalloc(&x_dev, 2*size_f->nx*sizeof(myReal)));
    cudaErrorHandle(hipMemcpy(x_dev, x, 2*size_f->nx*sizeof(myReal), hipMemcpyHostToDevice));
    
    myReal* Omega_dev;
    cudaErrorHandle(hipMalloc(&Omega_dev, 2*numR*size_f->nx*sizeof(myReal)));
    cudaErrorHandle(hipMemcpy(Omega_dev, Omega, 2*numR*size_f->nx*sizeof(myReal), hipMemcpyHostToDevice));
    
    myReal* invGd_dev;
    cudaErrorHandle(hipMalloc(&invGd_dev, 4*sizeof(myReal)));
    cudaErrorHandle(hipMemcpy(invGd_dev, invGd, 4*sizeof(myReal), hipMemcpyHostToDevice));

    myReal* fc_x2_dev;
    cudaErrorHandle(hipMalloc(&fc_x2_dev, size_f->nx*sizeof(myReal)));
    thrust::device_ptr<myReal> fc_x2_thr(fc_x2_dev);

    short* indx2 = (short*) malloc(size_f->nx*sizeof(short));
    for (int i = 0; i < size_f->nx; i++) {
        indx2[i] = i;
    }

    short* indx2_dev;
    cudaErrorHandle(hipMalloc(&indx2_dev, size_f->nx*sizeof(short)));
    cudaErrorHandle(hipMemcpy(indx2_dev, indx2, size_f->nx*sizeof(short), hipMemcpyHostToDevice));

    short* indx2_sort_dev;
    cudaErrorHandle(hipMalloc(&indx2_sort_dev, size_f->nx*sizeof(short)));
    thrust::device_ptr<short> indx2_sort_thr(indx2_sort_dev);

    myReal* fcL_dev;
    cudaErrorHandle(hipMalloc(&fcL_dev, nD*numR*size_f->nx*sizeof(myReal)));

    short* fcL_indx_dev;
    cudaErrorHandle(hipMalloc(&fcL_indx_dev, nD*numR*size_f->nx*sizeof(short)));

    myReal* fc_x2 = (myReal*) malloc(nD*sizeof(myReal));
    short* fc_indx2 = (short*) malloc(nD*sizeof(short));
    myReal* fc_x1x2 = (myReal*) malloc(size_f->nx*size_f->nx*sizeof(myReal));
    short* fc_numx1 = (short*) malloc(size_f->nx*sizeof(short));
    myReal* fc_normal = (myReal*) malloc(size_f->nx*sizeof(myReal));

    myReal* fc_normal_dev;
    cudaErrorHandle(hipMalloc(&fc_normal_dev, size_f->nx*sizeof(myReal)));

    for (int iR = 0; iR < numR; iR++) {
        memset(fc_numx1, 0, size_f->nx*sizeof(short));

        for (int ix1 = 0; ix1 < size_f->nx; ix1++) {
            // compute all densities for a given R and Omega^+
            get_fc <<<size_f->const_2Bx, size_f->const_2Bx>>> (fc_x2_dev, x_dev+2*ix1, Omega_dev+2*iR, invGd_dev, numR, c_normal);
            
            // find the largest nD densities
            cudaErrorHandle(hipMemcpy(indx2_sort_dev, indx2_dev, size_f->nx*sizeof(short), hipMemcpyDeviceToDevice));
            thrust::stable_sort_by_key(fc_x2_thr, fc_x2_thr+size_f->nx, indx2_sort_thr, thrust::greater<myReal>());

            cudaErrorHandle(hipMemcpy(fcL_dev+iR*nD+ix1*nD*numR, fc_x2_dev, nD*sizeof(myReal), hipMemcpyDeviceToDevice));
            cudaErrorHandle(hipMemcpy(fcL_indx_dev+iR*nD+ix1*nD*numR, indx2_sort_dev, nD*sizeof(short), hipMemcpyDeviceToDevice));

            cudaErrorHandle(hipMemcpy(fc_x2, fc_x2_dev, nD*sizeof(myReal), hipMemcpyDeviceToHost));
            cudaErrorHandle(hipMemcpy(fc_indx2, indx2_sort_dev, nD*sizeof(short), hipMemcpyDeviceToHost));

            // prepare for normalization
            for (int i = 0; i < nD; i++) {
                fc_x1x2[fc_numx1[fc_indx2[i]] + fc_indx2[i]*size_f->nx] = fc_x2[i];
                fc_numx1[fc_indx2[i]]++;
            }
        }

        // normalization
        for (int ix2 = 0; ix2 < size_f->nx; ix2++) {
            fc_normal[ix2] = thrust::reduce(fc_x1x2+ix2*size_f->nx, fc_x1x2+ix2*size_f->nx+fc_numx1[ix2]);
            fc_normal[ix2] = fc_normal[ix2]*dx2;
        }
        
        cudaErrorHandle(hipMemcpy(fc_normal_dev, fc_normal, size_f->nx*sizeof(myReal), hipMemcpyHostToDevice));
        get_fc_normal <<<size_f->nx, nD>>> (fcL_dev+iR*nD, fc_normal_dev, fcL_indx_dev+iR*nD, numR);

        printf("No. %d finished, total: %d\n", iR+1, numR);
    }

    // calculate fc*lambda
    dim3 blocksize_fcL(nD, size_f->const_2Bx, 1);
    dim3 gridsize_fcL(size_f->const_2Bx, numR, 1);

    myReal* lambda_dev;
    cudaErrorHandle(hipMalloc(&lambda_dev, numR*size_f->nx*sizeof(myReal)));
    cudaErrorHandle(hipMemcpy(lambda_dev, lambda, numR*size_f->nx*sizeof(myReal), hipMemcpyHostToDevice));

    get_fcL <<<gridsize_fcL, blocksize_fcL>>> (fcL_dev, lambda_dev, fcL_indx_dev, dx2);
    cudaErrorHandle(hipMemcpy(fcL, fcL_dev, nD*numR*size_f->nx*sizeof(myReal), hipMemcpyDeviceToHost));
    cudaErrorHandle(hipMemcpy(fcL_indx, fcL_indx_dev, nD*numR*size_f->nx*sizeof(short), hipMemcpyDeviceToHost));

    // free memory
    cudaErrorHandle(hipFree(x_dev));
    cudaErrorHandle(hipFree(Omega_dev));
    cudaErrorHandle(hipFree(invGd_dev));
    cudaErrorHandle(hipFree(fc_x2_dev));
    cudaErrorHandle(hipFree(indx2_dev));
    cudaErrorHandle(hipFree(indx2_sort_dev));
    cudaErrorHandle(hipFree(fcL_dev));
    cudaErrorHandle(hipFree(fcL_indx_dev));
    cudaErrorHandle(hipFree(lambda_dev));

    free(indx2);
    free(fc_x2);
    free(fc_indx2);
    free(fc_x1x2);
    free(fc_numx1);
    free(fc_normal);
}

__global__ void get_fc(myReal* fc_x2, const myReal* x, const myReal* Omega, const myReal* invGd, const int numR, const myReal c_normal)
{
    int indx = threadIdx.x + blockIdx.x*blockDim.x;
    int indOmega = 2*numR*indx;
    
    myReal dOmega[2];
    dOmega[0] = x[0] - Omega[indOmega];
    dOmega[1] = x[1] - Omega[indOmega+1];

    myReal fc_local = invGd[0]*dOmega[0]*dOmega[0] + (invGd[1]+invGd[2])*dOmega[0]*dOmega[1] + invGd[3]*dOmega[1]*dOmega[1];
    fc_local = myexp(-0.5*fc_local)*c_normal;

    fc_x2[indx] = fc_local;
}

__global__ void get_fc_normal(myReal* fc, const myReal* fc_normal, const short* fc_indx, const int numR)
{
    int ind_fc = threadIdx.x + blockIdx.x*blockDim.x*numR;
    int ind_normal = fc_indx[ind_fc];

    fc[ind_fc] = fc[ind_fc] / fc_normal[ind_normal];
}

__global__ void get_fcL(myReal* fcL, const myReal* lambda, const short* fcL_indx, const myReal dx2)
{
    int ind_fcL = threadIdx.x + blockIdx.y*blockDim.x + (threadIdx.y+blockIdx.x*blockDim.y)*blockDim.x*gridDim.y;
    int ind_lambda = blockIdx.y + fcL_indx[ind_fcL]*gridDim.y;

    fcL[ind_fcL] *= (lambda[ind_lambda]*dx2);
}

